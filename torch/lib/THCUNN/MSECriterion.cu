#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#if THRUST_PATH
    #include <thrust/fill.h>
    #include <thrust/functional.h>
    #include <thrust/device_ptr.h>
    #include <thrust/reduce.h>
    #include <thrust/inner_product.h>
    #if CUDA_VERSION >= 7000
        #include <thrust/system/cuda/execution_policy.h>
    #endif
#else
    #include <bolt/amp/functional.h>
    #include <bolt/amp/inner_product.h>
    #include <bolt/amp/iterator/ubiquitous_iterator.h>
#endif

struct mse_functor
{
  __host__ __device__ float operator()(const float &x, const float &y) const
  {
    float z = x-y;
    return z*z;
  }
};

void THNN_CudaMSECriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 2, input, target);
  THArgCheck(THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
    "input and target need to have the same number of elements"
  );

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  float sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, (float) 0,
    thrust::plus<float>(), mse_functor());
#else
  auto input_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  float sum = bolt::amp::inner_product(input_data,
                                       input_data+size,
                                       target_data, 0.0f,
                                       bolt::amp::plus<float>(),
                                       mse_functor());
#endif
  if (sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  THCudaTensor_set1d(state, output, 0, sum);
}

struct mse_updateGradInput_functor
{
  float norm;

  __host__ __device__
  explicit
  mse_updateGradInput_functor(float norm_)
    : norm(norm_)
  {}

  __host__ __device__
  float operator()(float x, float y) const
  {
    return norm * (x - y);
  }
};

void THNN_CudaMSECriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);
  THArgCheck(THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
    "input and target need to have the same number of elements"
  );

  long size = THCudaTensor_nElement(state, input);
  float norm = sizeAverage ? 2.f/size : 2.f;

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    mse_updateGradInput_functor(norm));
#else
  auto input_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  auto gradInput_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, gradInput));

  bolt::amp::transform(input_data,
                       input_data + size,
                       target_data,
                       gradInput_data,
                       mse_updateGradInput_functor(norm));
#endif

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
}
