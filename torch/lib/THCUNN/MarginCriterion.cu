#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#if THRUST_PATH
    #include <thrust/fill.h>
    #include <thrust/functional.h>
    #include <thrust/device_ptr.h>
    #include <thrust/reduce.h>
    #include <thrust/inner_product.h>
#else
    #include <bolt/amp/functional.h>
    #include <bolt/amp/inner_product.h>
    #include <bolt/amp/iterator/ubiquitous_iterator.h>
#endif

struct margin_functor
{
  __host__ __device__
  explicit
  margin_functor(float margin)
    : margin(margin)
  {}

  __host__ __device__
  float operator()(float x, float y) const
  {
    float z = margin - x * y;
    return z >= 0 ? z : 0;
  }

  float margin;
};

void THNN_CudaMarginCriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage, float margin)
{
  THCUNN_assertSameGPU(state, 2, input, target);

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  float sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), margin_functor(margin));
#else
  auto input_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  float sum = bolt::amp::inner_product(input_data,
                                       input_data + size,
                                       target_data,
                                       0.0f,
                                       bolt::amp::plus<float>(),
                                       margin_functor(margin));
#endif

  if (sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  THCudaTensor_set1d(state, output, 0, sum);
}

struct margin_updateGradInput_functor
{
  float margin, norm;

  __host__ __device__
  margin_updateGradInput_functor(float margin_, float norm_)
    : margin(margin_)
    , norm(norm_)
  {}

  __host__ __device__
  float operator()(float x, float y) const
  {
    return (x * y) < margin ? -norm * y : 0;
  }
};

void THNN_CudaMarginCriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage, float margin)
{
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  long size = THCudaTensor_nElement(state, input);
  float norm = sizeAverage ? 1.f/size : 1;

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, margin_updateGradInput_functor(margin, norm));
#else
  auto input_data = bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data = bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  auto gradInput_data = bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, gradInput));

  bolt::amp::transform(input_data,
                       input_data + size,
                       target_data,
                       gradInput_data,
                       margin_updateGradInput_functor(margin, norm));
#endif

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
}
