#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel( float *output, float *input, int classSize, int height, int width)
{
  int batchIndex = hipBlockIdx_x;
  int index = hipThreadIdx_x;

  while (index < height*width) {
    int y = index / width;
    int x = index % width;
    if (y >= height)
      break;

    // calculate input starting index in cuda layout (B x H x W x C)
    int inputStartIndex =
      (height*width*classSize)*batchIndex +
      (width*classSize)*y +
      (classSize)*x;

    float sum = 0;
    for (int i = 0; i < classSize; i++) {
#ifdef __HIP_PLATFORM_HCC__
      sum += expf(input[inputStartIndex + i]);
#else
      sum += __expf(input[inputStartIndex + i]);
#endif
    }
    sum = 1.0f / sum;

    for (int i = 0; i < classSize; i++) {
      // calculate output index in torch layout (B x C x H x W)
      int outputIndex =
        (classSize*height*width)*batchIndex +
        (height*width)*i +
        (width)*y +
        x;
#ifdef __HIP_PLATFORM_HCC__
      output[outputIndex] = logf(sum * expf(input[inputStartIndex + i]));
#else
      output[outputIndex] = logf(sum * __expf(input[inputStartIndex + i]));
#endif
    }
    index += hipBlockDim_x;
  }
}

__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel( float *gradInput, float *output, float *gradOutput, int classSize, int height, int width)
{
  int batchIndex = hipBlockIdx_x;
  int index = hipThreadIdx_x;

  while (index < height*width) {
    int y = index / width;
    int x = index % width;
    if (y >= height)
      break;

    // calculate output starting index in cuda layout (B x H x W x C)
    int outputStartIndex =
      (height*width*classSize)*batchIndex +
      (width*classSize)*y +
      (classSize)*x;

    float sum = 0;
    for (int i = 0; i < classSize; i++) {
      sum += gradOutput[outputStartIndex + i];
    }

    for (int i = 0; i < classSize; i++) {
      // calculate input index in torch layout (B x C x H x W)
      int inputIndex =
        (classSize*height*width)*batchIndex +
        (height*width)*i +
        (width)*y +
        x;
#ifdef __HIP_PLATFORM_HCC__
      gradInput[inputIndex] = gradOutput[outputStartIndex + i] - expf(output[outputStartIndex + i]) * sum;
#else
      gradInput[inputIndex] = gradOutput[outputStartIndex + i] - __expf(output[outputStartIndex + i]) * sum;
#endif
    }
    index += hipBlockDim_x;
  }
}

struct MaxFloat
{
  __device__ __forceinline__ float operator()(float max, float v) const
  {
    return fmaxf(max, v);
  }
};

struct SumFloat
{
  __device__ __forceinline__ float operator()(float sum, float v) const
  {
    return sum + v;
  }
};

struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(float v)
    : max_k(v)
  {}

  __device__ __forceinline__ float operator()(float sum, float v) const
  {
#ifdef __HIP_PLATFORM_HCC__
    return sum + expf(v - max_k);
#else
    return sum + expf(v - max_k);
#endif
  }

  const float max_k;
};

struct NoFinal
{
  __device__ __forceinline__ float operator()(float v) const
  {
    return v;
  }
};

struct LSMFinal
{
  __device__ __forceinline__ LSMFinal(float m)
    : max_k(m)
  {}

  __device__ __forceinline__ float operator()(float v) const
  {
    return max_k + logf(v);
  }

  const float max_k;
};

template <typename Reduction, typename Finalize>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal,
            const Finalize& f)
{
  // To avoid RaW races from chaining blockReduce calls together, we
  // need a sync here
  __syncthreads();

  smem[hipThreadIdx_x] = val;

  __syncthreads();

  float warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if ((hipThreadIdx_x / 32) == 0) // only threads in warp1 go into this (if)
  {
    int lane = hipThreadIdx_x % 32; // from 0 to 31

    // if less than 1024 threads per block, then only activate the relevant lanes
    if (lane < hipBlockDim_x / 32)
    {
#pragma unroll
      for (int i = 0; i < 32; ++i)
      {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }

      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  float blockVal = defaultVal;

  if (hipThreadIdx_x == 0)
  {
    for (int i = 0; i < hipBlockDim_x / 32; ++i)
    {
      blockVal = r(blockVal, smem[i]);
    }

    smem[0] = f(blockVal);
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <typename Reduction>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal)
{
  return blockReduce<Reduction, NoFinal>(smem, val, r, defaultVal, NoFinal());
}

template <typename Reduction, int ILP>
__device__ __forceinline__ float
ilpReduce(float* data,
          int size,
          const Reduction& r,
          float defaultVal)
{
  float threadVal = defaultVal;
  int offset = hipThreadIdx_x;

  int last = size % (ILP * hipBlockDim_x);

  // Body (unroll by ILP times)
  for (; offset < size - last; offset += hipBlockDim_x * ILP)
  {
    float tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmp[j] = data[offset + j * hipBlockDim_x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      threadVal = r(threadVal, tmp[j]);
    }
  }

  // Epilogue
  for (; offset < size; offset += hipBlockDim_x)
  {
    threadVal = r(threadVal, data[offset]);
  }

  return threadVal;
}

template <int ILP>
__global__ void
cunn_LogSoftMax_updateOutput_kernel( float *output, float *input, int classes)
{
  //HIP_DYNAMIC_SHARED( float, buffer)
  __shared__ float buffer[1024];
  // forward pointers to batch[hipBlockIdx_x]
  // each block handles a sample in the mini-batch
  input += hipBlockIdx_x * classes;
  output += hipBlockIdx_x * classes;

  // find the max of the batch
  float threadMax =
    ilpReduce<MaxFloat, ILP>(input, classes, MaxFloat(), -FLT_MAX);
  // find the max over all batches
  float max_k =
    blockReduce<MaxFloat>(buffer, threadMax, MaxFloat(), -FLT_MAX);

  float threadExp =
    ilpReduce<SumExpFloat, ILP>(input, classes, SumExpFloat(max_k), 0.0f);
  float logsum_k =
    blockReduce<SumFloat, LSMFinal>(
      buffer, threadExp, SumFloat(), 0.0f, LSMFinal(max_k));

  // Output LSM (hand ILP)
  int offset = hipThreadIdx_x;

  int last = classes % (ILP * hipBlockDim_x);
  for (; offset < classes - last; offset += hipBlockDim_x * ILP)
  {
    float tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * hipBlockDim_x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      output[offset + j * hipBlockDim_x] = tmp[j] - logsum_k;
    }
  }

  for (; offset < classes; offset += hipBlockDim_x)
  {
    output[offset] = input[offset] - logsum_k;
  }
}

template <int ILP>
__global__ void
cunn_LogSoftMax_updateGradInput_kernel( float *gradInput,
                                       float *output,
                                       float *gradOutput,
                                       int classes)
{
  //HIP_DYNAMIC_SHARED( float, buffer)
  __shared__ float buffer[1024];
  gradInput += hipBlockIdx_x * classes;
  output += hipBlockIdx_x * classes;
  gradOutput += hipBlockIdx_x * classes;

  float threadSum =
    ilpReduce<SumFloat, 4>(gradOutput, classes, SumFloat(), 0.0f);
  float sum_k =
    blockReduce<SumFloat>(buffer, threadSum, SumFloat(), 0.0f);

  // Update gradInput (hand ILP)
  int offset = hipThreadIdx_x;
  int last = classes % (ILP * hipBlockDim_x);
  for (; offset < classes - last; offset += hipBlockDim_x * ILP)
  {
    float tmpGradOutput[ILP];
    float tmpOutput[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmpGradOutput[j] = gradOutput[offset + j * hipBlockDim_x];
      tmpOutput[j] = output[offset + j * hipBlockDim_x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      gradInput[offset + j * hipBlockDim_x] =
#ifdef __HIP_PLATFORM_HCC__
        tmpGradOutput[j] - expf(tmpOutput[j]) * sum_k;
#else
        tmpGradOutput[j] - __expf(tmpOutput[j]) * sum_k;
#endif
    }
  }

  for (; offset < classes; offset += hipBlockDim_x)
  {
    gradInput[offset] =
#ifdef __HIP_PLATFORM_HCC__
      gradOutput[offset] - expf(output[offset]) * sum_k;
#else
      gradOutput[offset] - __expf(output[offset]) * sum_k;
#endif
  }
}

void THNN_CudaLogSoftMax_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  THCudaTensor_resizeAs(state, output, input);

  bool spatial  = false;
  int batchSize = 1;
  int classSize = 0;
  int height = 0;
  int width = 0;

  int ndims = THCudaTensor_nDimension(state, input);

  if (ndims == 1)
  {
    classSize = THCudaTensor_size(state, input, 0);
    input = THCudaTensor_newContiguous(state, input);
  }
  else if (ndims == 2)
  {
    batchSize = THCudaTensor_size(state, input, 0);
    classSize = THCudaTensor_size(state, input, 1);
    input = THCudaTensor_newContiguous(state, input);
  }
  else if (ndims == 3)
  {
    spatial = true;
    classSize = THCudaTensor_size(state, input, 0);
    height = THCudaTensor_size(state, input, 1);
    width = THCudaTensor_size(state, input, 2);

    // create contiguous tensor with cuda layout from tensor with torch layout
    // C x H x W -> W x H x C
    THCudaTensor_transpose(state, input, input, 0, 2);
    // W x H x C -> H x W x C
    THCudaTensor_transpose(state, input, input, 0, 1);
    THCudaTensor *transposedInput = THCudaTensor_newContiguous(state, input);
    THCudaTensor_transpose(state, input, input, 0, 1);
    THCudaTensor_transpose(state, input, input, 0, 2);
    input = transposedInput;
  }
  else if (ndims == 4)
  {
    spatial = true;
    batchSize = THCudaTensor_size(state, input, 0);
    classSize = THCudaTensor_size(state, input, 1);
    height = THCudaTensor_size(state, input, 2);
    width = THCudaTensor_size(state, input, 3);

    // create contiguous tensor with cuda layout from tensor with torch layout
    // B x C x H x W -> B x W x H x C
    THCudaTensor_transpose(state, input, input, 1, 3);
    // B x W x H x C -> B x H x W x C
    THCudaTensor_transpose(state, input, input, 1, 2);
    THCudaTensor *transposedInput = THCudaTensor_newContiguous(state, input);
    THCudaTensor_transpose(state, input, input, 1, 2);
    THCudaTensor_transpose(state, input, input, 1, 3);
    input = transposedInput;
  }
  else
  {
    THError("1D, 2D, 3D or 4D Tensor expected");
  }

  if (!spatial)
  {
    dim3 grid(batchSize);
    dim3 block(1024);

    hipLaunchKernelGGL((cunn_LogSoftMax_updateOutput_kernel<2>), dim3(grid), dim3(block), block.x * sizeof(float), THCState_getCurrentStream(state), 
        THCudaTensor_data(state, output),
        THCudaTensor_data(state, input),
        classSize
    );
  }
  else
  {
    dim3 grid(batchSize);
    dim3 block(1024);

    hipLaunchKernelGGL((cunn_SpatialLogSoftMax_updateOutput_kernel), dim3(grid), dim3(block), 0, THCState_getCurrentStream(state), 
        THCudaTensor_data(state, output),
        THCudaTensor_data(state, input),
        classSize, height, width
    );
  }

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
  {
    THError(hipGetErrorString(errcode));
  }

  THCudaTensor_free(state, input);
}

void THNN_CudaLogSoftMax_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput,
  THCudaTensor *gradInput, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  THCudaTensor_resizeAs(state, gradInput, output);

  bool spatial  = false;
  int batchSize = 1;
  int classSize = 0;
  int height = 0;
  int width = 0;

  int ndims = THCudaTensor_nDimension(state, input);

  if (ndims == 1)
  {
    classSize = THCudaTensor_size(state, gradInput, 0);
    output = THCudaTensor_newContiguous(state, output);
    gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  }
  else if (ndims == 2)
  {
    batchSize = THCudaTensor_size(state, gradInput, 0);
    classSize = THCudaTensor_size(state, gradInput, 1);
    output = THCudaTensor_newContiguous(state, output);
    gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  }
  else if (ndims == 3)
  {
    spatial = true;
    classSize = THCudaTensor_size(state, input, 0);
    height = THCudaTensor_size(state, input, 1);
    width = THCudaTensor_size(state, input, 2);

    // create contiguous tensor with cuda layout from tensor with torch layout
    // C x H x W -> W x H x C
    THCudaTensor_transpose(state, output, output, 0, 2);
    // W x H x C -> H x W x C
    THCudaTensor_transpose(state, output, output, 0, 1);
    THCudaTensor *transposedOutput = THCudaTensor_newContiguous(state, output);
    THCudaTensor_transpose(state, output, output, 0, 1);
    THCudaTensor_transpose(state, output, output, 0, 2);
    output = transposedOutput;

    // create contiguous tensor with cuda layout from tensor with torch layout
    // C x H x W -> W x H x C
    THCudaTensor_transpose(state, gradOutput, gradOutput, 0, 2);
    // W x H x C -> H x W x C
    THCudaTensor_transpose(state, gradOutput, gradOutput, 0, 1);
    THCudaTensor *transposedGradOutput = THCudaTensor_newContiguous(state, gradOutput);
    THCudaTensor_transpose(state, gradOutput, gradOutput, 0, 1);
    THCudaTensor_transpose(state, gradOutput, gradOutput, 0, 2);
    gradOutput = transposedGradOutput;
  }
  else if (ndims == 4)
  {
    spatial = true;
    batchSize = THCudaTensor_size(state, gradInput, 0);
    classSize = THCudaTensor_size(state, input, 1);
    height = THCudaTensor_size(state, input, 2);
    width = THCudaTensor_size(state, input, 3);

    // create contiguous tensor with cuda layout from tensor with torch layout
    // B x C x H x W -> B x W x H x C
    THCudaTensor_transpose(state, output, output, 1, 3);
    // B x W x H x C -> B x H x W x C
    THCudaTensor_transpose(state, output, output, 1, 2);
    THCudaTensor *transposedOutput = THCudaTensor_newContiguous(state, output);
    THCudaTensor_transpose(state, output, output, 1, 2);
    THCudaTensor_transpose(state, output, output, 1, 3);
    output = transposedOutput;

    // create contiguous tensor with cuda layout from tensor with torch layout
    // B x C x H x W -> B x W x H x C
    THCudaTensor_transpose(state, gradOutput, gradOutput, 1, 3);
    // B x W x H x C -> B x H x W x C
    THCudaTensor_transpose(state, gradOutput, gradOutput, 1, 2);
    THCudaTensor *transposedGradOutput = THCudaTensor_newContiguous(state, gradOutput);
    THCudaTensor_transpose(state, gradOutput, gradOutput, 1, 2);
    THCudaTensor_transpose(state, gradOutput, gradOutput, 1, 3);
    gradOutput = transposedGradOutput;
  }
  else
  {
    THError("1D, 2D, 3D or 4D Tensor expected");
  }

  if (!spatial)
  {
    dim3 grid(batchSize);
    dim3 block(1024);

    hipLaunchKernelGGL((cunn_LogSoftMax_updateGradInput_kernel<2>), dim3(grid), dim3(block), block.x * sizeof(float), THCState_getCurrentStream(state), 
        THCudaTensor_data(state, gradInput),
        THCudaTensor_data(state, output),
        THCudaTensor_data(state, gradOutput),
        classSize
    );
  }
  else
  {
    dim3 grid(batchSize);
    dim3 block(1024);

    hipLaunchKernelGGL((cunn_SpatialLogSoftMax_updateGradInput_kernel), dim3(grid), dim3(block), 0, THCState_getCurrentStream(state), 
        THCudaTensor_data(state, gradInput),
        THCudaTensor_data(state, output),
        THCudaTensor_data(state, gradOutput),
        classSize, height, width
    );
  }

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
  {
    THError(hipGetErrorString(errcode));
  }

  THCudaTensor_free(state, gradOutput);
  THCudaTensor_free(state, output);
}
