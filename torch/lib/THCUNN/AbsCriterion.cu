#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#if THRUST_PATH
    #include <thrust/fill.h>
    #include <thrust/functional.h>
    #include <thrust/device_ptr.h>
    #include <thrust/reduce.h>
    #include <thrust/inner_product.h>
#else
    #include <bolt/amp/functional.h>
    #include <bolt/amp/inner_product.h>
    #include <bolt/amp/iterator/ubiquitous_iterator.h>
#endif

struct abs_functor
{
  __host__ __device__
  abs_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
  {
    float z = x-y;
    return z >= 0 ? z : -z;
  }

  __host__ __device__
  ~abs_functor() {}
};

void THNN_CudaAbsCriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 2, input, target);

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  float sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), abs_functor());
#else
  auto input_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  float sum = bolt::amp::inner_product(input_data,
                                       input_data+size,
                                       target_data, 0.0f,
                                       bolt::amp::plus<float>(),
                                       abs_functor());
#endif

  if (sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  THCudaTensor_set1d(state, output, 0, sum);
}

struct abs_updateGradInput_functor
{
  float norm;

  __host__ __device__
  abs_updateGradInput_functor() = default;

  __host__ __device__
  explicit abs_updateGradInput_functor(float norm_)
    : norm(norm_)
  {}

  abs_updateGradInput_functor(const abs_updateGradInput_functor& fun) = default;
  __host__ __device__
  float operator()(const float& x, const float& y) const
  {
    return (x - y) >= 0 ? norm : -norm;
  }
};

void THNN_CudaAbsCriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  long size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

#if THRUST_PATH
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor(norm));
#else
  auto input_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, input));
  auto target_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, target));
  auto gradInput_data =
      bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, gradInput));

  bolt::amp::transform(input_data,
                       input_data+size,
                       target_data,
                       gradInput_data,
                       abs_updateGradInput_functor(norm));
#endif

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
}
