#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCHalf.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCNumerics.cuh"

template <typename T>
struct TensorAddConstantOp {
  __host__ __device__
  explicit
  TensorAddConstantOp(T v) : val(v) {}

  __device__ __forceinline__
  void operator()(T* out, T* in) { *out = *in + val; }

  __device__ __forceinline__
  void operator()(T* v) { *v += val; }

  __host__ __device__
  ~TensorAddConstantOp() {}

  T val;
};

#ifdef CUDA_HALF_TENSOR
    template <>
    struct TensorAddConstantOp<half> {
    #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
      __host__ __device__
      explicit
      TensorAddConstantOp(half v) : val(v) {}
    #else
      __host__
      explicit
      TensorAddConstantOp(half v) : fval(THC_half2float(v)) {}
    #endif

      __device__ __forceinline__
      void operator()(half* out, half* in)
      {
        #if defined(__HIP_PLATFORM_HCC__)
          *out = *in + val;
        #elif defined(CUDA_HALF_INSTRUCTIONS)
            *out = __hadd(*in, val);
        #else
          float fin = __half2float(*in);
          float fout = fin + fval;
          *out = __float2half(fout);
        #endif
      }

      __device__ __forceinline__
      void operator()(half* v)
      {
        #if defined(__HIP_PLATFORM_HCC__)
           *v += val;
        #elif defined(CUDA_HALF_INSTRUCTIONS)
          *v = __hadd(*v, val);
        #else
          float fv = __half2float(*v);
          fv += fval;
          *v = __float2half(fv);
        #endif
      }

    #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
      half val;
    #else
      float fval;
    #endif
    };
#endif // CUDA_HALF_TENSOR


template <typename T>
struct TensorSubConstantOp {
  __host__ __device__
  explicit
  TensorSubConstantOp(T v) : val(v) {}

  __device__ __forceinline__
  void operator()(T* out, T* in) { *out = *in - val; }

  __device__ __forceinline__
  void operator()(T* v) { *v -= val; }

  __host__ __device__
  ~TensorSubConstantOp() {}

  T val;
};


#ifdef CUDA_HALF_TENSOR
    template <>
    struct TensorSubConstantOp<half> {
      #if defined(__HIP_PLATFORM_HCC__)
        __host__ __device__
        explicit
        TensorSubConstantOp(half v) : val{v} {}
      #elif defined(CUDA_HALF_INSTRUCTIONS)
        __host__ __device__
        explicit
        TensorSubConstantOp(half v)
          : val(THC_float2half(-(THC_half2float(v)))) {}
      #else
        __host__
        explicit
        TensorSubConstantOp(half v): fval(-(THC_half2float(v))) {}
      #endif

      __device__ __forceinline__
      void operator()(half* out, half* in)
      {
        #if defined(__HIP_PLATFORM_HCC__)
          *out = *in + val;
        #elif defined(CUDA_HALF_INSTRUCTIONS)
          *out = __hadd(*in, val);
        #else
          float fin = __half2float(*in);
          float fout = fin + fval;
          *out = __float2half(fout);
        #endif
      }

      __device__ __forceinline__
      void operator()(half* v)
      {
        #if defined(__HIP_PLATFORM_HCC__)
          *v += val;
        #elif defined(CUDA_HALF_INSTRUCTIONS)
          *v = __hadd(*v, val);
        #else
          float fv = __half2float(*v);
          fv += fval;
          *v = __float2half(fv);
        #endif
      }

    #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
      half val;
    #else
      float fval;
    #endif
    };
#endif // CUDA_HALF_TENSOR


template <typename T>
struct TensorMulConstantOp {
  __host__ __device__
  explicit
  TensorMulConstantOp(T v) : val(v) {}

  __device__ __forceinline__
  void operator()(T* out, T* in) { *out = *in * val; }

  __device__ __forceinline__
  void operator()(T* v) { *v *= val; }

  __host__ __device__
  ~TensorMulConstantOp() {}

  T val;
};

#ifdef CUDA_HALF_TENSOR
    template <>
    struct TensorMulConstantOp<half> {
      #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
        __host__ __device__
        explicit
        TensorMulConstantOp(half v) : val(v) {}
      #else
        explicit
        TensorMulConstantOp(half v) : fval(THC_half2float(v)) {}
      #endif

        __device__ __forceinline__
        void operator()(half* out, half* in)
        {
          #if defined(__HIP_PLATFORM_HCC__)
            *out = *in * val;
          #elif defined(CUDA_HALF_INSTRUCTIONS)
            *out = __hmul(*in, val);
          #else
            float fin = __half2float(*in);
            float fout = fin * fval;
            *out = __float2half(fout);
          #endif
        }

        __device__ __forceinline__
        void operator()(half* v)
        {
          #if defined(__HIP_PLATFORM_HCC__)
            *v = *v * val;
          #elif defined(CUDA_HALF_INSTRUCTIONS)
            *v = __hmul(*v, val);
          #else
            float fv = __half2float(*v);
            fv *= fval;
            *v = __float2half(fv);
          #endif
        }

        #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
          half val;
        #else
          float fval;
        #endif
    };
#endif // CUDA_HALF_TENSOR

template <typename T>
struct TensorDivConstantOp {
  __host__ __device__
  explicit
  TensorDivConstantOp(T v) : val(v) {}
  __device__ __forceinline__
  void operator()(T* out, T* in) { *out = *in / val; }

  __device__ __forceinline__
  void operator()(T* v) { *v /= val; }

  __host__ __device__
  ~TensorDivConstantOp() {}

  T val;
};

template <>
struct TensorDivConstantOp<float> {
  __host__ __device__
  explicit
  TensorDivConstantOp(float v) : val(1.f / v) {}
  __device__ __forceinline__
  void operator()(float* out, float* in) { *out = *in * val; }

  __device__ __forceinline__
  void operator()(float* v) { *v *= val; }

  __host__ __device__
  ~TensorDivConstantOp() {}

  float val;
};

template <>
struct TensorDivConstantOp<double> {
  __host__ __device__
  explicit
  TensorDivConstantOp(double v) : val(1. / v) {}

  __device__ __forceinline__
  void operator()(double* out, double* in) { *out = *in * val; }

  __device__ __forceinline__
  void operator()(double* v) { *v *= val; }

  __host__ __device__
  ~TensorDivConstantOp() {}

  double val;
};

#ifdef CUDA_HALF_TENSOR
  template <>
  struct TensorDivConstantOp<half> {
    #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
      __host__ __device__
      explicit
      TensorDivConstantOp(half v) : val(ScalarInv<half>::to(v)) {}
    #else
      TensorDivConstantOp(half v) : fval(1.f / THC_half2float(v)) {}
    #endif
    __device__ __forceinline__
    void operator()(half* out, half* in)
    {
      #if defined(__HIP_PLATFORM_HCC__)
        *out = *in * val;
      #elif defined(CUDA_HALF_INSTRUCTIONS)
        *out = __hmul(*in, val);
      #else
        float fin = __half2float(*in);
        float fout = fin * fval;
        *out = __float2half(fout);
      #endif
    }

    __device__ __forceinline__
    void operator()(half* v)
    {
      #if defined(__HIP_PLATFORM_HCC__)
        *v *= val;
      #elif defined(CUDA_HALF_INSTRUCTIONS)
        *v = __hmul(*v, val);
      #else
        float fv = __half2float(*v);
        fv *= fval;
        *v = __float2half(fv);
      #endif
    }

      #if defined(CUDA_HALF_INSTRUCTIONS) || defined(__HIP_PLATFORM_HCC__)
        half val;
      #else
        float fval;
      #endif
  };
#endif // CUDA_HALF_TENSOR

template <int Upper>
struct TensorTriOp {
  __host__ __device__
  TensorTriOp(float *start_, long stride0_, long stride1_, long k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_)
  {}

  __device__ __forceinline__
  int mask(float *in)
  {
    ptrdiff_t n = in - start;
    long row, col;
    if (stride0 > stride1)
    {
      row = (long) (n / stride0);
      col = (long) ((n % stride0) / stride1);
    }
    else
    {
      row = (long) ((n % stride1) / stride0);
      col = (long) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__
  void operator()(float* out, float* in) { *out = mask(in) ? *in : 0; }

  __device__ __forceinline__
  void operator()(float* v) { if (!mask(v)) *v = 0; }

  float *start;
  long stride0, stride1, k;
};

void THCudaTensor_tril(THCState *state, THCudaTensor *self_, THCudaTensor *src_, long k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<0> op(start, stride0, stride1, k);
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_triu(THCState *state, THCudaTensor *self_, THCudaTensor *src_, long k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}

#include "generic/THCTensorMathPairwise.cu"
#include "THCGenerateAllTypes.h"

// Copy the kth diagonal of a matrix B to a vector A.
__global__
void THCudaTensor_copyFromDiagonal(
    float* a,
    float* b,
    ptrdiff_t start,
    ptrdiff_t size,
    ptrdiff_t strideSum,
    ptrdiff_t strideA)
{
  for (ptrdiff_t linearIndex = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
       linearIndex < size;
       linearIndex += hipGridDim_x * hipBlockDim_x) {
    const ptrdiff_t bOffset = start + strideSum * linearIndex;
    a[strideA * linearIndex] = b[bOffset];
  }
}

// Copy vector B to the kth diagonal of a matrix A
__global__
void THCudaTensor_copyToDiagonal(
    float* a,
    float* b,
    ptrdiff_t start,
    ptrdiff_t size,
    ptrdiff_t strideSum,
    ptrdiff_t strideB)
{
  for (ptrdiff_t linearIndex = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
       linearIndex < size;
       linearIndex += hipGridDim_x * hipBlockDim_x) {
    const ptrdiff_t aOffset = start + strideSum * linearIndex;
    a[aOffset] = b[strideB * linearIndex];
  }
}

void THCudaTensor_diag(
    THCState *state, THCudaTensor *self_, THCudaTensor *src_, long k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  int nDimension = THCudaTensor_nDimension(state, src_);
  THArgCheck(
      (nDimension == 2) || (nDimension == 1),
      1,
      "expected a matrix or a vector");
  if (nDimension == 2) {
    long stride0 = THCudaTensor_stride(state, src_, 0);
    long stride1 = THCudaTensor_stride(state, src_, 1);
    long size0 = THCudaTensor_size(state, src_, 0);
    long size1 = THCudaTensor_size(state, src_, 1);
    long size =
        (k > 0) ? min((long long)size0, (long long)size1 - k)
                : min((long long)size0 + k, (long long)size1);
    THCudaTensor_resize1d(state, self_, size);
    long strideSelf = THCudaTensor_stride(state, self_, 0);
    const dim3 threads(min(
        (long long)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock,
        (long long)size));
    dim3 grid(min((long long)1024, (long long)THCCeilDiv(size, (long)threads.x)));
    long start = (k >= 0 ? k * stride1 : -k * stride0);
    hipLaunchKernelGGL(
        THCudaTensor_copyFromDiagonal,
        dim3(grid),
        dim3(threads),
        0,
        THCState_getCurrentStream(state),
        THCudaTensor_data(state, self_),
        THCudaTensor_data(state, src_),
        start,
        size,
        stride0 + stride1,
        strideSelf);
  } else {
    ptrdiff_t totalElements = THCudaTensor_nElement(state, src_);
    ptrdiff_t size = (k > 0) ? totalElements + k : totalElements - k;
    long strideSrc = THCudaTensor_stride(state, src_, 0);
    THCudaTensor_resize2d(state, self_, size, size);
    THCudaTensor_zero(state, self_);
    long stride0 = THCudaTensor_stride(state, self_, 0);
    long stride1 = THCudaTensor_stride(state, self_, 1);
    const dim3 threads(min(
        (long long)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock,
        (long long)size));
    dim3 grid(min((long long)1024, (long long)THCCeilDiv(size, (ptrdiff_t)threads.x)));
    ptrdiff_t start = (k >= 0 ? k * stride1 : -k * stride0);
    hipLaunchKernelGGL(
        THCudaTensor_copyToDiagonal,
        dim3(grid),
        dim3(threads),
        0,
        THCState_getCurrentStream(state),
        THCudaTensor_data(state, self_),
        THCudaTensor_data(state, src_),
        start,
        totalElements,
        stride0 + stride1,
        strideSrc);
  }
  THCudaCheck(hipGetLastError());
}

float THCudaTensor_trace(THCState *state, THCudaTensor *src_)
{
  THAssert(THCudaTensor_checkGPU(state, 1, src_));
  THArgCheck((src_->nDimension == 2), 1, "expected a matrix");
  THCudaTensor *diag = THCudaTensor_new(state);
  THCudaTensor_diag(state, diag, src_, 0);
  float trace = THCudaTensor_sumall(state, diag);
  THCudaTensor_free(state, diag);
  return trace;
}
