#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"
#include "THCReduceApplyUtils.cuh"
#ifdef CURAND_PATH
  #include <hiprand.h>
  #include <hiprand/hiprand_kernel.h>
  #include <hiprand/hiprand_mtgp32_host.h>
  #include <rocrand/rocrand_mtgp32_11213.h>
#else
  #include <hip/hip_hcc.h>
  #include "MTGP/hiprand_mtgp32.h"
#endif

#ifdef THRUST_PATH
    #include <thrust/functional.h>
#else
    #include <bolt/amp/functional.h>
#endif

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

/* Sets up generator. Allocates but does not create the generator states. */
void initializeGenerator(THCState *state, Generator* gen)
{
#ifdef CURAND_PATH
  THCudaCheck(THCudaMalloc(state, (void**)&gen->gen_states, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  THCudaCheck(THCudaMalloc(state, (void**)&gen->kernel_params, sizeof(mtgp32_kernel_params)));
#else
  assert(gen);
  gen->h_gen_states = new HipRandStateMtgp32;
  assert(gen->h_gen_states);
  hipStream_t currentStream = THCState_getCurrentStream(state);
  hc::accelerator_view* current_accl_view;
  hipHccGetAcceleratorView(currentStream, &current_accl_view);
  HipRandStateMtgp32_init(*current_accl_view, gen->h_gen_states);
#endif
}

/* Frees memory allocated during setup. */
void destroyGenerator(THCState *state, Generator* gen)
{
#ifdef CURAND_PATH
  if (gen->gen_states)
  {
    THCudaCheck(THCudaFree(state, gen->gen_states));
    gen->gen_states = NULL;
  }
  if (gen->kernel_params)
  {
    THCudaCheck(THCudaFree(state, gen->kernel_params));
    gen->kernel_params = NULL;
  }
#else
  if (gen->h_gen_states) {
    HipRandStateMtgp32_release(gen->h_gen_states);
    delete gen->h_gen_states;
  }
#endif
}

/* Creates a new generator state given the seed. */
void createGeneratorState(THCState* state, Generator* gen, unsigned long seed)
{
#ifdef CURAND_PATH
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
  if (hiprandMakeMTGP32KernelState(gen->gen_states, mtgp32dc_params_fast_11213,
                                  gen->kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
#else
  hipStream_t currentStream = THCState_getCurrentStream(state);
  hc::accelerator_view* current_accl_view;
  hipHccGetAcceleratorView(currentStream, &current_accl_view);

  if (mtgp32_init_params_kernel(*current_accl_view, mtgp32_params_fast_11213, gen->h_gen_states)) {
    THError("Creating MTGP constants failed.");
  }

  // Using device API
  if (mtgp32_init_seed_kernel(*current_accl_view, gen->h_gen_states, seed)) {
    THError("Creating MTGP kernel state failed.");
  }

#endif
}

/* Initialize generator array (must be called before any other function) */
void THCRandom_init(THCState* state, int devices, int current_device)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  rng_state->num_devices = devices;
  rng_state->gen = (Generator*)malloc(rng_state->num_devices * sizeof(Generator));
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    rng_state->gen[i].initf = 0;
    rng_state->gen[i].initial_seed = 0;
#ifdef CURAND_PATH
    rng_state->gen[i].gen_states = NULL;
    rng_state->gen[i].kernel_params = NULL;
#else
    rng_state->gen[i].h_gen_states = NULL;
#endif
  }
}

/* Destroy generators and free memory */
void THCRandom_shutdown(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  if (rng_state->gen == NULL) return;
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    destroyGenerator(state, &rng_state->gen[i]);
  }
  free(rng_state->gen);
  rng_state->gen = NULL;
}

/* Manually set the generator seed */
static void THCRandom_manualSeedGen(THCState* state, Generator* gen, unsigned long seed)
{
  gen->initial_seed = seed;
  createGeneratorState(state, gen, seed);
  gen->initf = 1;
}

/* Get the generator for the current device */
Generator* THCRandom_getGenerator(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);

  int device;
  THCudaCheck(hipGetDevice(&device));
  if (device >= rng_state->num_devices) THError("Invalid device index.");

  Generator* gen = &rng_state->gen[device];
  if (gen->initf == 0)
  {
    initializeGenerator(state, gen);
    THCRandom_manualSeedGen(state, gen, (unsigned long)time(0));
  }
  return gen;
}

#ifdef CURAND_PATH
struct hiprandStateMtgp32* THCRandom_generatorStates(struct THCState* state)
{
  return THCRandom_getGenerator(state)->gen_states;
}
#else
struct HipRandStateMtgp32* THCRandom_generatorStates(struct THCState* state)
{
  return THCRandom_getGenerator(state)->h_gen_states;
}
#endif
/* Random seed */
unsigned long THCRandom_seed(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeed(state, s);
  return s;
}

unsigned long THCRandom_seedAll(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeedAll(state, s);
  return s;
}

/* Manually set the seed */
void THCRandom_manualSeed(THCState* state, unsigned long seed)
{
  Generator* gen = THCRandom_getGenerator(state);
  THCRandom_manualSeedGen(state, gen, seed);
}

void THCRandom_manualSeedAll(THCState* state, unsigned long seed)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  int currentDevice;
  THCudaCheck(hipGetDevice(&currentDevice));
  for (int i = 0; i < rng_state->num_devices; ++i) {
    THCudaCheck(hipSetDevice(i));
    THCRandom_manualSeed(state, seed);
  }
  THCudaCheck(hipSetDevice(currentDevice));
}

/* Get the initial seed */
unsigned long THCRandom_initialSeed(THCState* state)
{
  return THCRandom_getGenerator(state)->initial_seed;
}

void THCRandom_getRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

  // The RNG state comprises the MTPG32 states and the seed.
#ifdef CURAND_PATH
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
#else
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(HipRandStateMtgp32);
#endif
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
#ifdef CURAND_PATH
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->gen_states,
                         states_size, hipMemcpyDeviceToHost));
#else
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->h_gen_states,
                         states_size, hipMemcpyDeviceToHost));
#endif
  memcpy(THByteTensor_data(rng_state) + states_size, &gen->initial_seed, seed_size);
}

#ifdef CURAND_PATH
__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
  state[hipThreadIdx_x].k = kernel;
}
#else

#endif

void THCRandom_setRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

#ifdef CURAND_PATH
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
#else
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(HipRandStateMtgp32);
#endif
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");

#ifdef CURAND_PATH
  THCudaCheck(hipMemcpy(gen->gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  hipLaunchKernelGGL(
    set_rngstate_kernel,
    dim3(1),
    dim3(MAX_NUM_BLOCKS),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    gen->kernel_params);
#else
  THCudaCheck(hipMemcpy(gen->h_gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
#endif
  memcpy(&gen->initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
}

#ifdef CURAND_PATH
#define GENERATE_KERNEL1(NAME, ARG1, CURAND_FUNC, TRANSFORM)                   \
__global__ void NAME(hiprandStateMtgp32 *state, int size, float *result, ARG1)  \
{                                                                              \
  int idx = hipBlockIdx_x * BLOCK_SIZE + hipThreadIdx_x;                             \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                     \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {      \
    float x = CURAND_FUNC(&state[hipBlockIdx_x]);                                 \
    if (i < size) {                                                            \
      x = TRANSFORM;                                                           \
      result[i] = x;                                                           \
    }                                                                          \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, ARG1, ARG2, CURAND_FUNC, TRANSFORM)                   \
__global__ void NAME(hiprandStateMtgp32 *state, int size, float *result, ARG1, ARG2)  \
{                                                                                    \
  int idx = hipBlockIdx_x * BLOCK_SIZE + hipThreadIdx_x;                                   \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                           \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {            \
    float x = CURAND_FUNC(&state[hipBlockIdx_x]);                                       \
    if (i < size) {                                                                  \
      x = TRANSFORM;                                                                 \
      result[i] = x;                                                                 \
    }                                                                                \
  }                                                                                  \
}
#else

#define GENERATE_KERNEL1(NAME, ARG1, HIPRAND_FUNC, FUNCTOR)                   \
void NAME(THCState* state, HipRandStateMtgp32 *rngstate, int size, float *result, ARG1)  \
{ \
  hipStream_t currentStream = THCState_getCurrentStream(state); \
  hc::accelerator_view* current_accl_view; \
  hipHccGetAcceleratorView(currentStream, &current_accl_view); \
  HIPRAND_FUNC##_kernel(*current_accl_view, rngstate, result, FUNCTOR); \
}

#define GENERATE_KERNEL2(NAME, ARG1, ARG2, HIPRAND_FUNC, FUNCTOR)                   \
void NAME(THCState* state, HipRandStateMtgp32 *rngstate, int size, float *result, ARG1, ARG2)  \
{                                                                                    \
  hipStream_t currentStream = THCState_getCurrentStream(state); \
  hc::accelerator_view* current_accl_view; \
  hipHccGetAcceleratorView(currentStream, &current_accl_view); \
  HIPRAND_FUNC##_kernel(*current_accl_view, rngstate, result, FUNCTOR);                                                 \
}

#endif
#ifdef CURAND_PATH
GENERATE_KERNEL2(generate_uniform, double a, double b, hiprand_uniform, x * (b-a) + a)
GENERATE_KERNEL1(generate_bernoulli, double p, hiprand_uniform, (float)x <= p)
GENERATE_KERNEL2(generate_normal, double mean, double stdv, hiprand_normal, (x * stdv) + mean)
GENERATE_KERNEL1(generate_geometric, double p, hiprand_uniform, (log(1-x) / log(p)) + 1)
GENERATE_KERNEL1(generate_exponential, double lambda, hiprand_uniform, (float)(-1. / lambda * log(1-x)))
GENERATE_KERNEL2(generate_cauchy, double median, double sigma, hiprand_uniform, (float)(median + sigma * tan(M_PI*(x-0.5))))
#else

// Adding All HC based constructors

class user_uniform_functor {
  double _a;
  double _b;
public:
  __host__ __device__
  user_uniform_functor(double a, double b) : _a(a), _b(b) {}

  __host__ __device__
  double operator()(float x) const { return x * (_b - _a) + _a; }
};


class user_bernoulli_functor {
  double _p;
public:
  __host__ __device__
  explicit
  user_bernoulli_functor(double p) : _p(p) {}

  __host__ __device__
  double operator()(float x) const { return static_cast<double>(x) <= _p; }
};


class user_normal_functor {
  double _stdv;
  double _mean;
public:
  __host__ __device__
  user_normal_functor(double stdv, double mean) : _stdv(stdv), _mean(mean) {}

  __host__ __device__
  double operator()(float x) const { return (x * _stdv) + _mean; }
};

class user_geometric_functor {
  double _p;
public:
  __host__ __device__
  explicit
  user_geometric_functor(double p) : _p(p) {}

  __device__
  double operator()(float x) const
  {
      return (log((double)(1 - x)) / log(_p)) + 1;
  }
};

class user_exponential_functor {
  double _lambda;
public:
  __host__ __device__
  explicit
  user_exponential_functor(double lambda) : _lambda(lambda) {}

  __device__
  double operator()(float x) const
  {
    return (double)(-1. / _lambda * log((double)(1 - x)));
  }
};

class user_cauchy_functor {
  double _median;
  double _sigma;
public:
  __host__ __device__
  user_cauchy_functor(double median, double sigma)
      : _median(median), _sigma(sigma)
  {}

  __device__
  double operator()(float x) const
  {
    return (double)(_median + _sigma * tan((double)M_PI * (x - 0.5)));
  }
};


GENERATE_KERNEL2(generate_uniform, double a, double b, user_uniform, user_uniform_functor(a, b))
GENERATE_KERNEL1(generate_bernoulli, double p, user_uniform, user_bernoulli_functor(p))
GENERATE_KERNEL2(generate_normal, double mean, double stdv, user_normal, user_normal_functor(stdv,  mean))
GENERATE_KERNEL1(generate_geometric,  double p, user_uniform, user_geometric_functor(p))
GENERATE_KERNEL1(generate_exponential, double lambda, user_uniform, user_exponential_functor(lambda))
GENERATE_KERNEL2(generate_cauchy, double median, double sigma, user_uniform, user_cauchy_functor(median, sigma))
#endif

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2

#ifdef CURAND_PATH
/* Separate kernel because hiprand_log_normal gets extra parameters. */
__global__ void generate_log_normal(hiprandStateMtgp32 *state, int size, float *result, float mean, float stddev)
{
  int idx = hipBlockIdx_x * BLOCK_SIZE + hipThreadIdx_x;
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {
    float x = hiprand_log_normal(&state[hipBlockIdx_x], mean, stddev);
    if (i < size) {
      result[i] = x;
    }
  }
}
#else
/* Separate kernel because hiprand_log_normal gets extra parameters. */
void generate_log_normal(THCState* state, HipRandStateMtgp32 *rngstate, int size, float* result, float mean, float stddev) {
  hipStream_t currentStream = THCState_getCurrentStream(state);
  hc::accelerator_view* current_accl_view;
  hipHccGetAcceleratorView(currentStream, &current_accl_view);
  user_log_normal_kernel(*current_accl_view, rngstate, result, mean, stddev);
}
#endif

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)
THC_API void THCudaTensor_uniform(THCState* state, THCudaTensor *self_, double a, double b)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_uniform,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    a,
    b);
  #else
    generate_uniform(state, gen->h_gen_states, size, data, a, b);
  #endif

  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_bernoulli(THCState* state, THCudaTensor *self_, double p)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_bernoulli,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    p);
  #else
  generate_bernoulli(state, gen->h_gen_states, size, data, p);
  #endif
  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_normal(THCState* state, THCudaTensor *self_, double mean, double stdv)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_normal,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    mean,
    stdv);
  #else
  generate_normal(state, gen->h_gen_states, size, data, mean, stdv);
  #endif

  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_logNormal(THCState* state, THCudaTensor *self_, double mean, double stdv)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_log_normal,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    mean,
    stdv);
  #else
    generate_log_normal(state, gen->h_gen_states, size, data, mean, stdv);
  #endif
  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_geometric(THCState* state, THCudaTensor *self_, double p)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_geometric,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    p);
  #else
  generate_geometric(state, gen->h_gen_states, size, data, p);
  #endif

  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_exponential(THCState* state, THCudaTensor *self_, double lambda)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_exponential,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    lambda);
  #else
  generate_exponential(state, gen->h_gen_states, size, data, lambda);
  #endif

  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_cauchy(THCState* state, THCudaTensor *self_, double median, double sigma)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);
  #ifdef CURAND_PATH
  hipLaunchKernelGGL(
    generate_cauchy,
    dim3(NUM_BLOCKS),
    dim3(BLOCK_SIZE),
    0,
    THCState_getCurrentStream(state),
    gen->gen_states,
    size,
    data,
    median,
    sigma);
  #else
  generate_cauchy(state, gen->h_gen_states, size, data, median, sigma);
  #endif

  THCudaTensor_freeCopyTo(state, self, self_);
};

__device__ int binarySearchForMultinomial(float* dist,
                                          int size,
                                          float val) {
  int start = 0;
  int end = size;

  while (end - start > 0) {
    int mid = start + (end - start) / 2;

    float midVal = dist[mid];
    if (midVal < val) {
      start = mid + 1;
    } else {
      end = mid;
    }
  }

  if (start == size) {
    // No probability mass or precision problems; just return the
    // first element
    start = 0;
  }

  return start;
}

// Normalizes the L1 norm of every row to 1; used by multinomial
__global__ void renormRowsL1(float* dist, long rows, long cols) {
  HIP_DYNAMIC_SHARED( float, smem)

  for (long row = hipBlockIdx_x; row < rows; row += hipGridDim_x) {
    float sum = 0.0f;
    for (long col = hipThreadIdx_x; col < cols; col += hipBlockDim_x) {
      sum += dist[row * cols + col];
    }

   sum = reduceBlock(reinterpret_cast<float*>(smem),
                      hipBlockDim_x,
                      sum,
    #if defined(THRUST_PATH)
                      thrust::plus<float>(),
    #else
                      bolt::amp::plus<float>(),
    #endif
                      0.0f);
    if (hipThreadIdx_x == 0) {
      smem[0] = sum;
    }
    __syncthreads();

    sum = smem[0];
    if (sum > 0.0f) {
      for (long col = hipThreadIdx_x; col < cols; col += hipBlockDim_x) {
        dist[row * cols + col] /= sum;
      }
    }
  }
}

void THCudaTensor_renormRows(struct THCState* state,
                             THCudaTensor* t) {
  THAssert(THCudaTensor_nDimension(state, t) == 2);
  long rows = THCudaTensor_size(state, t, 0);
  long cols = THCudaTensor_size(state, t, 1);

  hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  hipLaunchKernelGGL(
    renormRowsL1,
    dim3(grid),
    dim3(block),
    block.x * sizeof(float),
    THCState_getCurrentStream(state),
    THCudaTensor_data(state, t),
    rows,
    cols);
}

__global__
void sampleMultinomialOnce(
  float* dest, long distributions, int categories, float* dist)
{
  HIP_DYNAMIC_SHARED( float, smem)

  for (long curDist = hipBlockIdx_x;
       curDist < distributions; curDist += hipGridDim_x) {
    // Each block handles one distribution
    // First pass, find the total sum of the distribution
    float sum = 0.0f;
    for (int cat = hipThreadIdx_x; cat < categories; cat += hipBlockDim_x) {
      sum += dist[curDist * categories + cat];
    }

    // hipThreadIdx_x == 0 has the sum value from this
    sum = reduceBlock(reinterpret_cast<float*>(smem),
                      hipBlockDim_x,
                      sum,
    #if defined(THRUST_PATH)
                      thrust::plus<float>(),
    #else
                      bolt::amp::plus<float>(),
    #endif
                      0.0f);

    // Broadcast sum and sample value
    if (hipThreadIdx_x == 0) {
      smem[0] = sum;
      smem[1] = dest[curDist];
    }
    __syncthreads();

    sum = smem[0];
    float sample = smem[1];
    __syncthreads();

    if (sum == 0.0f || sample == 0.0f) {
      // Choose the first element
      if (hipThreadIdx_x == 0) {
        dest[curDist] = 1;
      }

      continue;
    }

    int chunks = THCCeilDiv(categories, (int) hipBlockDim_x);
    float prevHighProb = 0.0f;

    for (int chunk = 0; chunk < chunks; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * hipBlockDim_x + hipThreadIdx_x;

      float val =
        cat < categories ? dist[curDist * categories + cat] / sum : 0.0f;
      smem[hipThreadIdx_x] = val;
      __syncthreads();

      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < hipBlockDim_x; offset *= 2) {
        float val = 0.0f;

        if (hipThreadIdx_x >= offset) {
          val = smem[hipThreadIdx_x - offset] + smem[hipThreadIdx_x];
        }

        __syncthreads();
        if (hipThreadIdx_x >= offset) {
          smem[hipThreadIdx_x] = val;
        }
        __syncthreads();
      }

      // Each thread will check to see if the sample falls in its
      // bucket
      float curBucket =
        smem[hipThreadIdx_x] + prevHighProb;
      float prevBucket =
        hipThreadIdx_x == 0 ? prevHighProb : smem[hipThreadIdx_x - 1] + prevHighProb;
      bool inBucket =
        (cat < categories) && (sample <= curBucket) && (sample > prevBucket);

      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        // FIXME: broadcast exit flag?
        dest[curDist] = cat + TH_INDEX_BASE;
      }

      // Store the previous scan's high value for future use
      prevHighProb += smem[hipBlockDim_x - 1];

      __syncthreads();
    }
  }
}

#ifdef CURAND_PATH
__global__ void
sampleMultinomialWithReplacement(hiprandStateMtgp32* state,
                                 int totalSamples,
                                 float* dest,
                                 long distributions,
                                 int categories,
                                 float* normDistPrefixSum) {
#else
__global__ void
sampleMultinomialWithReplacement(HipRandStateMtgp32* state,
                                 int totalSamples,
                                 float* dest,
                                 long distributions,
                                 int categories,
                                 float* normDistPrefixSum) {

#endif
  // At the moment, each warp computes one sample value in the binary
  // search due to divergence. It seems possible to compute multiple
  // values and limit divergence though later on. However, no matter
  // what, all block threads must participate in the hiprand_uniform
  // call to update the generator state.

  // The block determines the distribution for which we generate a point
  for (long curDist = hipBlockIdx_x;
       curDist < distributions;
       curDist += hipGridDim_x) {
    for (int sampleBase = 0;
         sampleBase < totalSamples; sampleBase += hipBlockDim_y) {
      // The warp determines the sample
      int sample = sampleBase + hipThreadIdx_y;

      // All threads participate in this
      float r = 0.0f;
      #ifdef CURAND_PATH
      r = hiprand_uniform(&state[hipBlockIdx_x]);
      #else
     // r = _hiprand_uniform(&state[hipBlockIdx_x]);
      #endif

      if (hipThreadIdx_x == 0 && sample < totalSamples) {
        // Find the bucket that a uniform sample lies in
        int choice = binarySearchForMultinomial(
          normDistPrefixSum + curDist * categories,
          categories,
          r);

        // Torch indices are 1-based
        dest[curDist * totalSamples + sample] = (float) choice + (float)TH_INDEX_BASE;
      }
    }
  }
}

#ifdef CURAND_PATH
__global__ void
sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state,
                                    int totalSamples,
                                    int sample,
                                    float* dest,
                                    long distributions,
                                    int categories,
                                    float* origDist,
                                    float* normDistPrefixSum) {
#else
__global__ void
sampleMultinomialWithoutReplacement(HipRandStateMtgp32* state,
                                    int totalSamples,
                                    int sample,
                                    float* dest,
                                    long distributions,
                                    int categories,
                                    float* origDist,
                                    float* normDistPrefixSum) {
#endif
  // At the moment, each warp computes one sample value in the binary
  // search due to divergence. It seems possible to compute multiple
  // values and limit divergence though later on. However, no matter
  // what, all block threads must participate in the hiprand_uniform
  // call to update the generator state.

  // The block and warp determines the distribution for which we
  // generate a point
  for (long curDistBase = hipBlockIdx_x * hipBlockDim_y;
       curDistBase < distributions;
       curDistBase += hipGridDim_x * hipBlockDim_y) {
    // The warp determines the distribution
    long curDist = curDistBase + hipThreadIdx_y;

    // All threads must participate in this
    float r = 0.0f;
    #ifdef CURAND_PATH
    r = hiprand_uniform(&state[hipBlockIdx_x]);
    #endif

    if (hipThreadIdx_x == 0 && curDist < distributions) {
      // Find the bucket that a uniform sample lies in
      int choice = binarySearchForMultinomial(
        normDistPrefixSum + curDist * categories,
        categories,
        r);

      // Torch indices are 1-based
      dest[curDist * totalSamples + sample] = (float) choice + (float)TH_INDEX_BASE;

      // Without replacement, so update the original probability so it
      // is not considered a second time
      origDist[curDist * categories + choice] = 0.0f;
    }
  }
}

THC_API void THCudaTensor_multinomial(struct THCState *state,
                                      THCudaTensor *self,
                                      THCudaTensor *prob_dist,
                                      int n_sample,
                                      int with_replacement)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, prob_dist));
  Generator* gen = THCRandom_getGenerator(state);

  int inputSize = THCudaTensor_nDimension(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  long numDist =
    inputSize == 1 ? 1 : THCudaTensor_size(state, prob_dist, 0);
  long numCategoriesLong =
    inputSize == 1 ? THCudaTensor_size(state, prob_dist, 0) :
    THCudaTensor_size(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  // It is possible that prob_dist is non-contiguous
  THCudaTensor* probDistContig =
    THCudaTensor_newContiguous(state, prob_dist);

  // Restructure data for 2d
  if (inputSize == 1) {
    THCudaTensor_resize2d(state, probDistContig, 1, numCategories);
  }

  THCudaTensor_resize2d(state, self, numDist, n_sample);

  if (n_sample == 1) {
    // Optimized allocation-free implementation

    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into the
    // result memory. The device RNG is thread-limited
    THCudaTensor_uniform(state, self, 0.0, 1.0);

    hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
    THAssert(props != NULL);

    int numSM = props->multiProcessorCount;
    int maxThreads = props->maxThreadsPerBlock;

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    hipLaunchKernelGGL(
      sampleMultinomialOnce,
      dim3(grid),
      dim3(block),
      block.x * sizeof(float),
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, self),
      numDist,
      numCategories,
      THCudaTensor_data(state, probDistContig));
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCudaTensor* origDist = THCudaTensor_new(state);
    THCudaTensor_resizeAs(state, origDist, probDistContig);
    THCudaTensor_copy(state, origDist, probDistContig);

    THCudaTensor* normDist = THCudaTensor_new(state);
    THCudaTensor_resizeAs(state, normDist, probDistContig);

    THCudaTensor* prefixSum = THCudaTensor_new(state);

    // Renorm along rows
    THCudaTensor_copy(state, normDist, origDist);
    THCudaTensor_renormRows(state, normDist);

    // Prefix sum along rows
    THCudaTensor_cumsum(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

#ifdef CURAND_PATH
      hipLaunchKernelGGL(
        sampleMultinomialWithReplacement,
        dim3(grid),
        dim3(block),
        0,
        THCState_getCurrentStream(state),
        gen->gen_states,
        n_sample,
        THCudaTensor_data(state, self),
        numDist,
        numCategories,
        THCudaTensor_data(state, prefixSum));
#else
      hipLaunchKernelGGL(
          sampleMultinomialWithReplacement,
          dim3(grid),
          dim3(block),
          0,
          THCState_getCurrentStream(state),
          gen->h_gen_states,
          n_sample,
          THCudaTensor_data(state, self),
          numDist,
          numCategories,
          THCudaTensor_data(state, prefixSum));
#endif
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, 4L);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCudaTensor_copy(state, normDist, origDist);
          THCudaTensor_renormRows(state, normDist);

          // Prefix sum along rows
          THCudaTensor_cumsum(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
#ifdef CURAND_PATH
        hipLaunchKernelGGL(
          sampleMultinomialWithoutReplacement,
          dim3(grid),
          dim3(block),
          0,
          THCState_getCurrentStream(state),
          gen->gen_states,
          n_sample,
          sample,
          THCudaTensor_data(state, self),
          numDist,
          numCategories,
          THCudaTensor_data(state, origDist),
          THCudaTensor_data(state, prefixSum));
#else
        hipLaunchKernelGGL(
          (sampleMultinomialWithoutReplacement),
          dim3(grid),
          dim3(block),
          0,
          THCState_getCurrentStream(state),
          gen->h_gen_states,
          n_sample,
          sample,
          THCudaTensor_data(state, self),
          numDist, numCategories,
          THCudaTensor_data(state, origDist),
          THCudaTensor_data(state, prefixSum));
#endif
      }
    }

    THCudaTensor_free(state, prefixSum);
    THCudaTensor_free(state, normDist);
    THCudaTensor_free(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaTensor_resize1d(state, self, n_sample);

    // Unfortunately, if prob_dist is contiguous already,
    // newContiguous is not a private copy, so we have to restructure
    // this too, so as to not affect prob_dist
    THCudaTensor_resize1d(state, probDistContig, numCategories);
  }

  THCudaTensor_free(state, probDistContig);
}

#undef NUM_BLOCKS
