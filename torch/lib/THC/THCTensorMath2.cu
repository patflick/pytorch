#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"
#include "THCTensorMathReduce.cuh"
#include "THCTensorMathPointwise.cuh"

#ifdef THRUST_PATH
  #include <thrust/device_ptr.h>
  #include <thrust/transform_reduce.h>
  #include <thrust/functional.h>
  #include <thrust/inner_product.h>
  #if CUDA_VERSION >= 7000
    #include <thrust/system/cuda/execution_policy.h>
  #endif
#else
  #include <bolt/amp/functional.h>
  #include <bolt/amp/inner_product.h>
  #include <bolt/amp/iterator/ubiquitous_iterator.h>
#endif

struct TensorTPowOp {
  __host__ __device__
  explicit
  TensorTPowOp(float v) : val(v) {}

  __device__ __forceinline__
  void operator()(float* out, float* in) { *out = powf(val, *in); }

  __device__ __forceinline__
  void operator()(float* v) { *v = powf(val, *v); }

  float val;
};

void THCudaTensor_tpow(
    THCState *state, THCudaTensor *self_, float value, THCudaTensor *src)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorTPowOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);
    if (!THC_pointwiseApply2(state, self_, src, TensorTPowOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorATan2Op {
  __device__ __forceinline__
  void operator()(float* out, float* a, float* b) { *out = atan2f(*a, *b); }
};

void THCudaTensor_atan2(
    THCState *state, THCudaTensor *self_, THCudaTensor *tx, THCudaTensor *ty)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self_, tx, ty));
  THArgCheck(THCudaTensor_nElement(state, tx) ==
             THCudaTensor_nElement(state, ty), 3, "sizes do not match");
  THCudaTensor_resizeAs(state, self_, tx);
  if (!THC_pointwiseApply3(state, self_, tx, ty, TensorATan2Op())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

float THCudaTensor_dist(
    THCState *state, THCudaTensor *self, THCudaTensor *src, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));
  self = THCudaTensor_newContiguous(state, self);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  src = THCudaTensor_newContiguous(state, src);
  float result = 0;
#ifdef THRUST_PATH
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));
  thrust::device_ptr<float> src_data(THCudaTensor_data(state, src));

  result = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+size, src_data, (float) 0,
    thrust::plus<float>(), TensorDistOp<float>(value));

#else
    auto self_data =
        bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, self));
    auto src_data =
        bolt::amp::make_ubiquitous_iterator(THCudaTensor_data(state, src));

    result = bolt::amp::inner_product(
        self_data,
        self_data + size,
        src_data,
        0.0f,
        bolt::amp::plus<float>(),
        TensorDistOp<float>(value));
#endif

  THCudaTensor_free(state, src);
  THCudaTensor_free(state, self);

  return pow(result, (float)1.0/value);
}

void THCudaTensor_rand(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THAssert(THCudaTensor_checkGPU(state, 1, r_));
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_uniform(state, r_, 0, 1);
}

void THCudaTensor_randn(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THAssert(THCudaTensor_checkGPU(state, 1, r_));
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_normal(state, r_, 0, 1);
}
